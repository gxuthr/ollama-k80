#include "hip/hip_runtime.h"
#include "im2col.cuh"

template <typename T>
static  __global__ void im2col_kernel(
        const float * x, T * dst, int64_t batch_offset,
        int64_t offset_delta, int64_t IC, int64_t IW, int64_t IH, int64_t OH, int64_t OW, int64_t KW, int64_t KH, int64_t pelements, int64_t CHW,
        int s0, int s1, int p0, int p1, int d0, int d1) {
    __shared__ float tile[32][32];
    
    const int tx = threadIdx.x;
    const int ty = threadIdx.y;
    if (tx < 32 && ty < 32) {
        tile[ty][tx] = x[ty * IW + tx];
    }
    __syncthreads();

    const int64_t i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i >= pelements) {
        return;
    }

    const int64_t  ksize = OW * (KH > 1 ? KW : 1);
    const int64_t  kx = i / ksize;
    const int64_t  kd = kx * ksize;
    const int64_t  ky = (i - kd) / OW;
    const int64_t  ix = i % OW;

    const int64_t  oh = blockIdx.y;
    const int64_t  batch = blockIdx.z / IC;
    const int64_t  ic = blockIdx.z % IC;

    const int64_t iiw = ix * s0 + kx * d0 - p0;
    const int64_t iih = oh * s1 + ky * d1 - p1;

    const int64_t offset_dst =
        ((batch * OH + oh) * OW + ix) * CHW +
        (ic * (KW * KH) + ky * KW + kx);

    if (iih < 0 || iih >= IH || iiw < 0 || iiw >= IW) {
        dst[offset_dst] = 0.0f;
    } else {
        const int64_t offset_src = ic * offset_delta + batch * batch_offset;
        dst[offset_dst] = tile[iih][iiw];
    }
}

template <typename T>
static void im2col_cuda(const float * x, T* dst,
    int64_t IW, int64_t IH, int64_t OW, int64_t OH, int64_t KW, int64_t KH, int64_t IC,
    int64_t batch, int64_t batch_offset, int64_t offset_delta,
    int s0,int s1,int p0,int p1,int d0,int d1, hipStream_t stream) {
    const int parallel_elements = OW * KW * KH;
    const int num_blocks = (parallel_elements + CUDA_IM2COL_BLOCK_SIZE - 1) / CUDA_IM2COL_BLOCK_SIZE;
    dim3 block_nums(num_blocks, OH, batch * IC);
    im2col_kernel<<<block_nums, CUDA_IM2COL_BLOCK_SIZE, 0, stream>>>(x, dst, batch_offset, offset_delta, IC, IW, IH, OH, OW, KW, KH, parallel_elements, (IC * KH * KW), s0, s1, p0, p1, d0, d1);
}

static void im2col_cuda_f16(const float * x, half * dst,
    int64_t IW, int64_t IH, int64_t OW, int64_t OH, int64_t KW, int64_t KH, int64_t IC,
    int64_t batch, int64_t batch_offset, int64_t offset_delta,
    int s0,int s1,int p0,int p1,int d0,int d1, hipStream_t stream) {

    im2col_cuda<half>(x, dst, IW, IH, OW, OH, KW, KH, IC, batch, batch_offset, offset_delta, s0, s1, p0, p1, d0, d1, stream);
}

static void im2col_cuda_f32(const float * x, float * dst,
    int64_t IW, int64_t IH, int64_t OW, int64_t OH, int64_t KW, int64_t KH, int64_t IC,
    int64_t batch, int64_t batch_offset, int64_t offset_delta,
    int s0,int s1,int p0,int p1,int d0,int d1, hipStream_t stream) {

    im2col_cuda<float>(x, dst, IW, IH, OW, OH, KW, KH, IC, batch, batch_offset, offset_delta, s0, s1, p0, p1, d0, d1, stream);
}

// K80优化的im2col实现
template <typename T>
static __global__ void k80_optimized_im2col_kernel(
    const float* x,
    T* dst,
    const int64_t batch_size,
    const int64_t channels,
    const int64_t height,
    const int64_t width,
    const int64_t kernel_h,
    const int64_t kernel_w,
    const int64_t pad_h,
    const int64_t pad_w,
    const int64_t stride_h,
    const int64_t stride_w,
    const int64_t dilation_h,
    const int64_t dilation_w) {
    
    // 使用共享内存缓存输入数据
    __shared__ float shared_input[32][32];
    
    const int64_t thread_idx = threadIdx.x + blockIdx.x * blockDim.x;
    const int64_t output_h = (height + 2 * pad_h - (dilation_h * (kernel_h - 1) + 1)) / stride_h + 1;
    const int64_t output_w = (width + 2 * pad_w - (dilation_w * (kernel_w - 1) + 1)) / stride_w + 1;
    const int64_t output_size = output_h * output_w;
    
    // 协作加载到共享内存
    if(threadIdx.x < 32 && threadIdx.y < 32) {
        int h = blockIdx.y * 32 + threadIdx.y;
        int w = blockIdx.x * 32 + threadIdx.x;
        if(h < height && w < width) {
            shared_input[threadIdx.y][threadIdx.x] = x[h * width + w];
        }
    }
    __syncthreads();
    
    // 计算输出
    if(thread_idx < output_size) {
        const int64_t out_h = thread_idx / output_w;
        const int64_t out_w = thread_idx % output_w;
        
        for(int64_t c = 0; c < channels; ++c) {
            for(int64_t kh = 0; kh < kernel_h; ++kh) {
                for(int64_t kw = 0; kw < kernel_w; ++kw) {
                    const int64_t h = out_h * stride_h - pad_h + kh * dilation_h;
                    const int64_t w = out_w * stride_w - pad_w + kw * dilation_w;
                    
                    if(h >= 0 && h < height && w >= 0 && w < width) {
                        dst[((c * kernel_h + kh) * kernel_w + kw) * output_size + thread_idx] = 
                            shared_input[h - blockIdx.y * 32][w - blockIdx.x * 32];
                    } else {
                        dst[((c * kernel_h + kh) * kernel_w + kw) * output_size + thread_idx] = 0;
                    }
                }
            }
        }
    }
}

void ggml_cuda_op_im2col(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];
    const float * src1_d = (const float *)src1->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src1->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F16 || dst->type == GGML_TYPE_F32);

    const int32_t s0 = ((const int32_t*)(dst->op_params))[0];
    const int32_t s1 = ((const int32_t*)(dst->op_params))[1];
    const int32_t p0 = ((const int32_t*)(dst->op_params))[2];
    const int32_t p1 = ((const int32_t*)(dst->op_params))[3];
    const int32_t d0 = ((const int32_t*)(dst->op_params))[4];
    const int32_t d1 = ((const int32_t*)(dst->op_params))[5];

    const bool is_2D = ((const int32_t*)(dst->op_params))[6] == 1;

    const int64_t IC = src1->ne[is_2D ? 2 : 1];
    const int64_t IH = is_2D ? src1->ne[1] : 1;
    const int64_t IW =         src1->ne[0];

    const int64_t KH = is_2D ? src0->ne[1] : 1;
    const int64_t KW =         src0->ne[0];

    const int64_t OH = is_2D ? dst->ne[2] : 1;
    const int64_t OW =         dst->ne[1];

    const size_t  delta_offset = src1->nb[is_2D ? 2 : 1] / 4; // nb is byte offset, src is type float32
    const int64_t batch        = src1->ne[is_2D ? 3 : 2];
    const size_t  batch_offset = src1->nb[is_2D ? 3 : 2] / 4; // nb is byte offset, src is type float32

    if(dst->type == GGML_TYPE_F16) {
        im2col_cuda_f16(src1_d, (half *) dst_d, IW, IH, OW, OH, KW, KH, IC, batch, batch_offset, delta_offset, s0, s1, p0, p1, d0, d1, stream);
    } else {
        im2col_cuda_f32(src1_d, (float *) dst_d, IW, IH, OW, OH, KW, KH, IC, batch, batch_offset, delta_offset, s0, s1, p0, p1, d0, d1, stream);
    }
}
